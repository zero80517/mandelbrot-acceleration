#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.cuh"

__global__ void FillMandelbrotKernel(uint *const bits,
                                     const double scaleFactor, const double centerX, const double centerY,
                                     const int Limit, const int MaxIterations,
                                     const uint *colormap, const uint ColormapSize,
                                     bool *const allBlack)
{
    int halfWidth = blockDim.x / 2;
    int halfHeight = gridDim.x / 2;
    int y = -halfHeight + blockIdx.x;
    int x = -halfWidth + threadIdx.x;
    uint *scanLine = bits + (y + halfHeight) * (halfWidth * 2) + (x + halfWidth);

    double ay = centerY + (y * scaleFactor);

    double ax = centerX + (x * scaleFactor);
    double a1 = ax;
    double b1 = ay;
    int numIterations = 0;

    do {
        ++numIterations;
        double a2 = (a1 * a1) - (b1 * b1) + ax;
        double b2 = (2 * a1 * b1) + ay;
        if ((a2 * a2) + (b2 * b2) > Limit)
            break;

        ++numIterations;
        a1 = (a2 * a2) - (b2 * b2) + ax;
        b1 = (2 * a2 * b2) + ay;
        if ((a1 * a1) + (b1 * b1) > Limit)
            break;
    } while (numIterations < MaxIterations);

    if (numIterations < MaxIterations) {
        *scanLine = colormap[numIterations % ColormapSize];
        if (*allBlack == true) {
            *allBlack = false;
        }
    } else {
        *scanLine = 0xff000000u;
    }
}

// Helper function for using CUDA.
hipError_t FillMandelbrotWithCuda(uint *const bits, const int halfWidth, const int halfHeight,
                                   const double scaleFactor, const double centerX, const double centerY,
                                   const int Limit, const int MaxIterations,
                                   const uint *colormap, const uint ColormapSize,
                                   bool *const allBlack)
{
    uint *dev_bits = 0;
    uint *dev_colormap = 0;
    bool *dev_allBlack = 0;
    const int size = (halfWidth * 2) * (halfHeight * 2);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_bits, size * sizeof(uint));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_colormap, ColormapSize * sizeof(uint));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_allBlack, 1 * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_colormap, colormap, ColormapSize * sizeof(uint), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_allBlack, allBlack, 1 * sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    FillMandelbrotKernel<<<halfHeight * 2, halfWidth * 2>>>(dev_bits,
                                                            scaleFactor, centerX, centerY,
                                                            Limit, MaxIterations,
                                                            dev_colormap, ColormapSize,
                                                            dev_allBlack);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FillMandelbrotKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching FillMandelbrotKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(bits, dev_bits, size * sizeof(uint), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(bits, dev_allBlack, 1 * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_bits);
    hipFree(dev_colormap);
    hipFree(dev_allBlack);
    
    return cudaStatus;
}
